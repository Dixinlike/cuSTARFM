#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gdal.h"
#include "gdal_priv.h"
#include "gdalwarper.h"
#include <stdio.h>
//#include"math.h"
#include<iostream>
#include"fusion.h"
//#include"cuLayer.h"
//#include"PARAMETER.h"
#define num_thread 256
#define num_block 256
__global__ void blending_pairs(float *a,float *b,float *c,float *d,float *wei,int width,int height,int w,float A,float error_lm,float error_mm,int class_num)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	float r_LM,r_MM, r_center_LM,r_center_MM; 
	int row,column;
	int i,j;
	float sum1,sum2;
	float st=0.0;
	int judge;
	float dis;
	//float wei;
	float weih,result;
	int kk=0;
	int rmin,rmax,smin,smax;
	for(int kkk=Idx;kkk<width*height;kkk=kkk+num_thread*num_block)
	{
		result=0;
		judge=0;
		wei[kkk]=0;
		kk=0;
		sum1=0,sum2=0;
		row=kkk/width;
		column=kkk%width;
		//if(row==1)
		//	wei=0;
		r_center_LM =d[kkk]-b[kkk]+error_lm;
		r_center_MM=d[kkk]-c[kkk]+1.412*error_mm;
		if(column-w/2<=0)
				rmin=0;
			else
				rmin = column-w/2;

			if(column+w/2>=width-1)
				rmax = width-1;
			else
				rmax = column+w/2;

			if(row-w/2<=0)
				smin=0;
			else
				smin = row-w/2;

			if(row+w/2>=height-1)
				smax = height-1;
			else
				smax = row+w/2;
			for(i=smin;i<=smax;i++)
			{
				for(j=rmin;j<=rmax;j++)
				{
					sum1+=b[i*width+j]*b[i*width+j];
					sum2+=b[i*width+j];
				}
			}
			//if(column==30&&row==30)
			//	result=0;
			st=sqrt(sum1/(w*w)-(sum2/(w*w))*(sum2/(w*w)))/ class_num;
			for(i=smin;i<=smax;i++)
			{
				for(j=rmin;j<=rmax;j++)
				{
					if(fabs(b[kkk]-b[i*width+j])<st)
					{
						r_LM=d[i*width+j]-b[i*width+j];
						r_MM=d[i*width+j]-c[i*width+j];
						if((r_center_LM>0&&r_LM<r_center_LM)||(r_center_LM<0&&r_LM>r_center_LM))
						{
							if((r_center_MM>0&&r_MM<r_center_MM)||(r_center_MM<0&&r_MM>r_center_MM))
							{
								r_LM=fabs(r_LM)+0.0001;
								r_MM=fabs(r_MM)+0.0001;
								if(kkk==i*width+j)
									judge=1;
								dis=float((row-i)*(row-i)+(column-j)*(column-j));
								dis=sqrt(dis)/A+1.0;
								weih=1.0/(dis* r_LM*r_MM);
								wei[kkk]+=weih;
								result+=weih*(c[i*width+j]+b[i*width+j]-d[i*width+j]);
								kk++;
							}
						}
					}
				}
			}
			if(kk==0)
			{
				a[kkk]=abs(b[kkk]+c[kkk]-d[kkk])*1000;
				wei[kkk]=1000;

			}
			else
			{
				if(judge==0)
					{
					dis=1.0;
					r_LM=fabs(d[kkk]-b[kkk])+0.0001;
					r_MM=fabs(d[kkk]-c[kkk])+0.0001;
					weih=1.0/(dis* r_LM*r_MM);
					result+=weih*(b[kkk]+c[kkk]-d[kkk]);
					wei[kkk]+=weih;
				}
			a[kkk]=result;
			//if(a[kkk]<0)
			//	a[kkk]=(b[kkk]+c[kkk]-d[kkk]);
			}
	}

}
__global__ void blending(float *a,float *b,float *c,float *d,int width,int height,int w,float A,float error_lm,float error_mm,int class_num)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	float r_LM,r_MM, r_center_LM,r_center_MM; 
	int row,column;
	int i,j;
	float sum1,sum2;
	float st=0.0;
	int judge;
	float dis;
	float wei;
	float weih,result;
	int kk=0;
	int rmin,rmax,smin,smax;
	for(int kkk=Idx;kkk<width*height;kkk=kkk+num_thread*num_block)
	{
		result=0;
		judge=0;
		wei=0;
		kk=0;
		sum1=0,sum2=0;
		row=kkk/width;
		column=kkk%width;
		//if(row==1)
		//	wei=0;
		r_center_LM =d[kkk]-b[kkk]+error_lm;
		r_center_MM=d[kkk]-c[kkk]+1.412*error_mm;
		if(column-w/2<=0)
				rmin=0;
			else
				rmin = column-w/2;

			if(column+w/2>=width-1)
				rmax = width-1;
			else
				rmax = column+w/2;

			if(row-w/2<=0)
				smin=0;
			else
				smin = row-w/2;

			if(row+w/2>=height-1)
				smax = height-1;
			else
				smax = row+w/2;
			for(i=smin;i<=smax;i++)
			{
				for(j=rmin;j<=rmax;j++)
				{
					sum1+=b[i*width+j]*b[i*width+j];
					sum2+=b[i*width+j];
				}
			}
			//if(column==30&&row==30)
			//	result=0;
			st=sqrt(sum1/(w*w)-(sum2/(w*w))*(sum2/(w*w)))/ class_num;
			for(i=smin;i<=smax;i++)
			{
				for(j=rmin;j<=rmax;j++)
				{
					if(fabs(b[kkk]-b[i*width+j])<st)
					{
						r_LM=d[i*width+j]-b[i*width+j];
						r_MM=d[i*width+j]-c[i*width+j];
						if((r_center_LM>0&&r_LM<r_center_LM)||(r_center_LM<0&&r_LM>r_center_LM))
						{
							if((r_center_MM>0&&r_MM<r_center_MM)||(r_center_MM<0&&r_MM>r_center_MM))
							{
								r_LM=fabs(r_LM)+0.0001;
								r_MM=fabs(r_MM)+0.0001;
								if(kkk==i*width+j)
									judge=1;
								dis=float((row-i)*(row-i)+(column-j)*(column-j));
								dis=sqrt(dis)/A+1.0;
								weih=1.0/(dis* r_LM*r_MM);
								wei+=weih;
								result+=weih*(c[i*width+j]+b[i*width+j]-d[i*width+j]);
								kk++;
							}
						}
					}
				}
			}
			if(kk==0)
			{
				a[kkk]=abs(b[kkk]+c[kkk]-d[kkk]);
				wei=10000;

			}
			else
			{
				if(judge==0)
					{
					dis=1.0;
					r_LM=fabs(d[kkk]-b[kkk])+0.0001;
					r_MM=fabs(d[kkk]-c[kkk])+0.0001;
					weih=1.0/(dis* r_LM*r_MM);
					result+=weih*(b[kkk]+c[kkk]-d[kkk]);
					wei+=weih;
				}
			a[kkk]=result/wei;
			//if(a[kkk]<0)
			//	a[kkk]=(b[kkk]+c[kkk]-d[kkk]);
			}
	}

}
void runtest1(float *BufferLandsat_0,float * BufferModis_0,float *BufferModis_1,float *out,int height,int width,int win_size ,float L_err,float M_err,int classnum,int A)
{
	float *cuda_a, *cuda_b, *cuda_c, *cuda_d;
	hipMalloc((void**)&cuda_a, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_b, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_c, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_d, sizeof( float)*height*width);
	hipMemcpy(cuda_b, BufferLandsat_0, sizeof(float)* height*width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_c, BufferModis_1, sizeof(float)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_d, BufferModis_0, sizeof( float)* height*width, hipMemcpyHostToDevice);
	blending<<<num_block,num_thread>>>(cuda_a , cuda_b , cuda_c , cuda_d , width,height,win_size,A,L_err,M_err,classnum);
	hipMemcpy(out,cuda_a,height*width*sizeof(float),hipMemcpyDeviceToHost);
	//for(int i=0;i<400;i++)
	//	std::cout<<out[i]<<"  ";
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(cuda_d);
}
void runtest(float *BufferLandsat_0,float * BufferModis_0,float *BufferModis_1,float *out,int height,int width,int win_size ,float L_err,float M_err,int classnum,int A)
{
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*win_size*win_size*num_block*num_thread*2)/(sizeof(float)*4);
	int sub_height=maxnum/width-win_size;
	//sub_height=3000;
	int kk=0;
	int i,j;
	float *sub_BufferIn11,*sub_BufferIn22,*sub_BufferIn33,*sub_out;
	for(int heiht_all=0;heiht_all<height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=height-1; 
		int data_start,data_end;
		if(task_start-win_size/2<=0)
			data_start= 0;
		else
			data_start=task_start-win_size/2;
		if(task_end+win_size/2>=height-1)
			data_end=height-1;
		else
			data_end=task_end+win_size/2;
		int data_height=data_end-data_start+1;
		sub_BufferIn11=new float[data_height*width];
		sub_BufferIn22=new float[data_height*width];
		sub_BufferIn33=new float[data_height*width];
		sub_out=new float[data_height*width];
		int copy;
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<width;j++)
				{
					sub_BufferIn11[copy*width+j]=BufferLandsat_0[i*width+j];
					sub_BufferIn22[copy*width+j]=BufferModis_0[i*width+j];
					sub_BufferIn33[copy*width+j]=BufferModis_1[i*width+j];
				}
				copy++;
			}
		int current=task_start-data_start;
		int task_height=task_end-task_start+1;
		runtest1(sub_BufferIn11,sub_BufferIn22,sub_BufferIn33,sub_out,data_height,width,win_size,L_err,M_err,classnum,A);
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<width;j++)
				{
					out[i*width+j]=sub_out[current*width+j];
				}
				current++;
			}
		delete []sub_BufferIn11;
		delete []sub_BufferIn22;
		delete []sub_BufferIn33;
		delete []sub_out;
		kk++;
	}
}
void runtest1_pairs(float *BufferLandsat_0,float * BufferModis_0,float *BufferModis_1,float *out,float *weight,int height,int width,int win_size ,float L_err,float M_err,int classnum,int A)
{

	float *cuda_a, *cuda_b, *cuda_c, *cuda_d,*cuda_weight;
	hipMalloc((void**)&cuda_weight, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_a, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_b, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_c, sizeof( float)*height*width);
	hipMalloc((void**)&cuda_d, sizeof( float)*height*width);
	hipMemcpy(cuda_b, BufferLandsat_0, sizeof(float)* height*width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_c, BufferModis_1, sizeof(float)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_d, BufferModis_0, sizeof( float)* height*width, hipMemcpyHostToDevice);
	blending_pairs<<<num_block,num_thread>>>(cuda_a , cuda_b , cuda_c , cuda_d ,cuda_weight,width,height,win_size,A,L_err,M_err,classnum);
	hipMemcpy(out,cuda_a,height*width*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(weight,cuda_weight,height*width*sizeof(float),hipMemcpyDeviceToHost);
	//for(int i=0;i<400;i++)
	//	std::cout<<out[i]<<"  ";
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(cuda_d);
	hipFree(cuda_weight);
}
void runtest_pairs(float *BufferLandsat_0,float * BufferModis_0,float *BufferModis_1,float *out,float *weight,int height,int width,int win_size ,float L_err,float M_err,int classnum,int A)
{
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*win_size*win_size*num_block*num_thread*2)/(sizeof(float)*5);
	int sub_height=maxnum/width-win_size;
   // sub_height=3000;
	int kk=0;
	int i,j;
	float *sub_BufferIn11,*sub_BufferIn22,*sub_BufferIn33,*sub_out,*sub_weight;
	for(int heiht_all=0;heiht_all<height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=height-1; 
		int data_start,data_end;
		if(task_start-win_size/2<=0)
			data_start= 0;
		else
			data_start=task_start-win_size/2;
		if(task_end+win_size/2>=height-1)
			data_end=height-1;
		else
			data_end=task_end+win_size/2;
		int data_height=data_end-data_start+1;
		sub_BufferIn11=new float[data_height*width];
		sub_BufferIn22=new float[data_height*width];
		sub_BufferIn33=new float[data_height*width];
		sub_out=new float[data_height*width];
		sub_weight=new float[data_height*width];
		int copy;
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<width;j++)
				{
					sub_BufferIn11[copy*width+j]=BufferLandsat_0[i*width+j];
					sub_BufferIn22[copy*width+j]=BufferModis_0[i*width+j];
					sub_BufferIn33[copy*width+j]=BufferModis_1[i*width+j];
				}
				copy++;
			}
		int current=task_start-data_start;
		int task_height=task_end-task_start+1;
		runtest1_pairs(sub_BufferIn11,sub_BufferIn22,sub_BufferIn33,sub_out,sub_weight,data_height,width,win_size,L_err,M_err,classnum,A);
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<width;j++)
				{
					out[i*width+j]=sub_out[current*width+j];
					weight[i*width+j]=sub_weight[current*width+j];
				}
				current++;
			}
		delete []sub_BufferIn11;
		delete []sub_BufferIn22;
		delete []sub_BufferIn33;
		delete []sub_out;
		delete []sub_weight;
		kk++;
	}
}
void Re_fusion(const char * BufferIn0,const char * BufferIn1,const char * BufferIn2,const char * BufferOut,int win_size,int classnum,float L_err,float M_err,int A,int pf,int pc)
{
	GDALAllRegister();
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8","NO"); 
	GDALDataset *Landsat0 = (GDALDataset*) GDALOpen(BufferIn0,GA_ReadOnly);
	int width,height;
	width = Landsat0->GetRasterXSize();
	height = Landsat0->GetRasterYSize();
	//width=2000;
	//height=2000;
	float *BufferLandsat_0=new float[width*height];
	float *BufferModis_0=new float[width*height];
	float *BufferModis_1=new float[width*height];
	float *out=new float[width*height];
	GDALRasterBand* hInBand1 = Landsat0->GetRasterBand(pf);
	hInBand1->RasterIO(GF_Read,0,0,width,height,BufferLandsat_0,width,height,GDT_Float32,0,0);
	GDALDataset *MODIS0 = (GDALDataset*) GDALOpen(BufferIn1,GA_ReadOnly);
	 hInBand1 = MODIS0->GetRasterBand(pc);
	hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_0,width,height,GDT_Float32,0,0);
	GDALDataset *Modis_1 = (GDALDataset*) GDALOpen(BufferIn2,GA_ReadOnly);
	 hInBand1 = Modis_1->GetRasterBand(pc);
	hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_1,width,height,GDT_Float32,0,0);
	 GDALDataset *LandsatDs;
	char* driverName = "GTiff";
	GDALDriver *pDriver = (GDALDriver*)GDALGetDriverByName(driverName);
	LandsatDs = pDriver->Create(BufferOut,width,height,1,GDT_Float32,NULL);
	double* geos=new double[1];
	Landsat0->GetGeoTransform(geos);
	LandsatDs->SetGeoTransform(geos);
	LandsatDs->SetProjection(Landsat0->GetProjectionRef());
	runtest(BufferLandsat_0,BufferModis_0,BufferModis_1,out,height,width,win_size, L_err, M_err, classnum,A);
	GDALRasterBand* HOut = LandsatDs->GetRasterBand(1);
	HOut->RasterIO(GF_Write,0,0,width,height,out,width,height,GDT_Float32,0,0);
	GDALClose(Landsat0);
	GDALClose(MODIS0);
	GDALClose(Modis_1);
	GDALClose(LandsatDs);
	delete []BufferLandsat_0;
	delete []BufferModis_0;
	delete []BufferModis_1;
	delete []out;
}
void Re_fusion2(CuLayer *psensor,PARAMETER *par)
{
	int i,j,m,c;
	long now1 = clock();
	for(c=0;c<par->NUM_PREDICTIONS;c++)
	{
		psensor[2*(par->NUM_PAIRS+c)+1].resize(psensor[0].getWidth(),psensor[0].getHeight());
		if(par->NUM_PAIRS==1)
		{
			runtest(psensor[0].getData(),psensor[1].getData(),psensor[2*(par->NUM_PAIRS+c)].getData(),psensor[2*(par->NUM_PAIRS+c)+1].getData(),psensor[0].getHeight(),psensor[0].getWidth(),par->WIN_SIZE,par->L_ERR,par->M_ERR,par->class_num,par->A);
			//char* driverName = "GTiff";
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
		}
		else
		{
			float *weight=new float[psensor[0].getHeight()*psensor[0].getWidth()];
			float *weight_all=new float[psensor[0].getHeight()*psensor[0].getWidth()];
			float *single=new float[psensor[0].getHeight()*psensor[0].getWidth()];
			memset(single,0, sizeof(float)*psensor[0].getHeight()*psensor[0].getWidth());
			memset(weight_all,0, sizeof(float)*psensor[0].getHeight()*psensor[0].getWidth());
			for( i=0;i<par->NUM_PAIRS;i++)
			{
				runtest_pairs(psensor[i].getData(),psensor[i+par->NUM_PAIRS].getData(),psensor[2*(par->NUM_PAIRS+c)].getData(),psensor[2*(par->NUM_PAIRS+c)+1].getData(),weight,psensor[0].getHeight(),psensor[0].getWidth(),par->WIN_SIZE,par->L_ERR,par->M_ERR,par->class_num,par->A);
				for( m=0;m<psensor[0].getHeight();m++)
				{
					//	j=j;
					for( j=0;j<psensor[0].getWidth();j++)
					{
						single[m*psensor[0].getWidth()+j]+=psensor[2*(par->NUM_PAIRS+c)+1].getData()[m*psensor[0].getWidth()+j];
						weight_all[m*psensor[0].getWidth()+j]+=weight[m*psensor[0].getWidth()+j];
					}
				}

			}
			for( m=0;m<psensor[0].getHeight();m++)
			{
				for( j=0;j<psensor[0].getWidth();j++)
				{
					psensor[2*(par->NUM_PAIRS+c)+1].getData()[m*psensor[0].getWidth()+j]=single[m*psensor[0].getWidth()+j]/weight_all[m*psensor[0].getWidth()+j];
				}
			}
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
			delete []weight;
			delete []weight_all;
			delete []single;
		}
	}
	 printf("GPU����ʱ��Ϊ��%dms\n", int(((double)(clock() - now1)) / CLOCKS_PER_SEC * 1000));
}
